#include "hip/hip_runtime.h"
//this is a lite version of a GPU accelerated N-body simulation. Has to be run on an NVIDIA machine with CUDA enabled.
//the interaction is just gravitation
//the simulation trajectory is to be visualized in VMD
#include <cstdio>
#include <cstdlib>
#include <cmath>

#define N 9999  // number of bodies
#define MASS 0  // row in array for mass
#define X_POS 1 // row in array for x position
#define Y_POS 2 // row in array for y position
#define Z_POS 3 // row in array for z position
#define X_VEL 4 // row in array for x velocity
#define Y_VEL 5 // row in array for y velocity
#define Z_VEL 6 // row in array for z velocity
#define G 10    // "gravitational constant" (not really)

float dt = 0.05; // time interval

// each thread computes new position of one body
__global__ void nbody(float *dev_body, float dt) {
int i=threadIdx.x + blockIdx.x* blockDim.x;
int j;
if(i<N) 
{    // force calculation
	float Fx_dir;
 	float Fy_dir;
	float Fz_dir;
    // initialize forces to zero
      Fx_dir = 0.0;
      Fy_dir = 0.0;
      Fz_dir = 0.0; 

 	for(j=0;j<N&&j!=i;j++)
 	{ 
// force on body x due to all other bodies 
	float x_diff, y_diff, z_diff;

	  x_diff = dev_body[i*7+X_POS] - dev_body [j*7+X_POS];  // difference in x direction
	  y_diff = dev_body[i*7+Y_POS] - dev_body [j*7+Y_POS];  // difference in y direction
	  z_diff = dev_body[i*7+Z_POS] - dev_body [j*7+Z_POS];  // difference in z direction

	  // calculate distance (r)
	  float rr = (x_diff * x_diff + y_diff * y_diff + z_diff * z_diff);
	  float r = sqrt(rr);

	  // force between bodies i and x
	  float F = G * dev_body[i*7+MASS] * dev_body[j*7+MASS] / r;

	  // if sufficiently far away, gravitation force
	  if (r > 10.0) {
	    Fx_dir += -F * x_diff / r;  // resolve forces in x and y directions
	    Fy_dir += -F * y_diff / r;  // and accumulate forces
	    Fz_dir += -F * z_diff / r;  // 
	  } 
	  else if(r<=10.0&&r>0.5){//avoid extremely large acceleration due to long time interval
	    // if too close, anti-gravitational force
	    Fx_dir -= -F * x_diff / r;  // resolve forces in x and y directions
	    Fy_dir -= -F * y_diff / r;  // and accumulate forces
	    Fz_dir -= -F * z_diff / r;  // 
	  }
	  }


    // update postions and velocity in array
        
	// update velocities
	dev_body[i*7+X_VEL] += Fx_dir * dt / dev_body[i*7+MASS];
	dev_body[i*7+Y_VEL] += Fy_dir * dt / dev_body[i*7+MASS];
	dev_body[i*7+Z_VEL] += Fz_dir * dt / dev_body[i*7+MASS];
	// update positions
	dev_body[i*7+X_POS] += dev_body[i*7+X_VEL] * dt;
	dev_body[i*7+Y_POS] += dev_body[i*7+Y_VEL] * dt;
	dev_body[i*7+Z_POS] += dev_body[i*7+Z_VEL] * dt;
	}
 
}


int main(int argc, char **argv) {
  float *body; // host data array of bodies
  float *dev_body; // device data array of bodies

  int tmax = 0;

  if (argc != 2) {
    fprintf(stderr, "Format: %s { number of timesteps }\n", argv[0]);
    exit (-1);
  }

  tmax = atoi(argv[1]);

  // allocate memory size for the body
  int bodysize = N * 7 * sizeof(float);    
  body = (float *)malloc(bodysize);
  hipMalloc((void**) &dev_body, bodysize);

  // assign each body a random position
  for (int i = 0; i < N; i++) {
    body[i * 7 + MASS] =  i%1001?1:1000;//create several heavy regions
    body[i * 7 + X_POS] = (i%2?0.0:200.0)+drand48() * 50.0;//define two galaxis
    body[i * 7 + Y_POS] = drand48() * 50.0;
    body[i * 7 + Z_POS] = drand48() * 20.0;//a plate-like distribution
    body[i * 7 + X_VEL] = drand48() * 0.1/body[i * 7 + MASS];
    body[i * 7 + Y_VEL] = (i%2?-10:10)+drand48() * 1.0/body[i * 7 + MASS];//angular momentum
    body[i * 7 + Z_VEL] = drand48() * 0.1/body[i * 7 + MASS];
  }

  // print out initial positions in PDB format

  printf("MODEL %8d\n", 0);
  for (int i = 0; i < N; i++) {
	printf("%s%7d  %s %s %s%4d     %7.0f %7.0f %7.0f  %4.2f  %4.3f\n",
           "ATOM", i+1, "CA ", "GLY", "A", i+1, body[i * 7 + X_POS], body[i * 7 + Y_POS], body[i * 7 + Z_POS], 1.00, 0.00);
  }
  printf("TER\nENDMDL\n");
 // copy nbody info over to GPU
    hipMemcpy(dev_body, body, bodysize, hipMemcpyHostToDevice);

  // step through each time step
  for (int t = 0; t < tmax; t++) {
 
    dim3 blockDim(1024);
    dim3 gridDim((int)ceil(N*1.0 / blockDim.x));

    // run nbody calculation
    nbody<<<gridDim, blockDim>>>(dev_body, dt);
    hipDeviceSynchronize();
if(!(t%1))//change output frequency by the mod factor. help to determine the robusticity quickly
{
    // copy nbody info back to CPU
    hipMemcpy(body, dev_body, bodysize, hipMemcpyDeviceToHost);

    // print out positions in PDB format
    printf("MODEL %8d\n", t+1);
    for (int i = 0; i < N; i++) {
	printf("%s%7d  %s %s %s%4d     %7.0f %7.0f %7.0f  %4.2f  %4.3f\n",
               "ATOM", i+1, "CA ", "GLY", "A", i+1, body[i * 7 + X_POS], body[i * 7 + Y_POS], body[i * 7 + Z_POS], 1.00, 0.00);
    }
    printf("TER\nENDMDL\n");
   }

  }  // end of time period loop
  free(body);
  hipFree(body);
}
